
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void sale(int *GPU_arr, int *GPU_price,int *GPU_out){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	__shared__ int temp[4];
	temp[threadIdx.x] = GPU_arr[i] * GPU_price[threadIdx.x];
 	__syncthreads();
	if(threadIdx.x==0){
	float sum = 0;
	sum = temp[0]+temp[1]+temp[2]+temp[3];	
	GPU_out[blockIdx.x] = sum;	
}
}

int main(void){
 int days = 7;
 int items = 4;
 int arr[28] = {
	3,5,2,0,
	2,4,5,1,
	0,3,3,1,
	3,5,4,4,
	4,5,5,3,
	10,13,21,16,
	8,11,15,8
};
 float prices[4] = {29.99,14.99,9.99,24.99};
 float out[7];
 int *GPU_arr;
 int *GPU_price; 
 int *GPU_out;
 hipMalloc((void**)&GPU_arr,sizeof(int)*28);
 hipMalloc((void**)&GPU_price,sizeof(float)*4);
 hipMalloc((void**)&GPU_out,sizeof(float)*7);
 hipMemcpy(GPU_arr,arr,sizeof(int)*28,hipMemcpyHostToDevice);
 hipMemcpy(GPU_price,prices,sizeof(float)*4,hipMemcpyHostToDevice);

 sale<<<7,4>>>(GPU_arr,GPU_price,GPU_out);
 hipMemcpy(out,GPU_out,sizeof(float)*7,hipMemcpyDeviceToHost);
 //printf("%d",*c);
 
 //printf("%d,%d,%d,%d\n",c[0],c[1],c[2],c[3]);
 return 0;
 }
