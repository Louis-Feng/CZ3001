
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void vector_add(int *d_a, int *d_b, int *d_c, int n){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	d_c[i] = d_a[i] + d_b[i];
}

int main(void){
 printf("Hello, World - from CPU!\n");
 int a[4] = {22,13,16,5};
 int b[4] = {5,22,17,37};
 int c[4];
 int *d_a; 
 int *d_b;
 int *d_c;
 hipMalloc((void**)&d_a,sizeof(int)*4);
 hipMalloc((void**)&d_b,sizeof(int)*4);
 hipMalloc((void**)&d_c,sizeof(int)*4);
 hipMemcpy(d_a,a,sizeof(int)*4,hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,sizeof(int)*4,hipMemcpyHostToDevice);
 //cudaMemcpy(d_c,c,sizeof(int)*4,cudaMemcpyHostToDevice);
 vector_add<<<1,4>>>(d_a,d_b,d_c,4);
 hipMemcpy(c,d_c,sizeof(int)*4,hipMemcpyDeviceToHost);
 
 printf("%d,%d,%d,%d\n",c[0],c[1],c[2],c[3]);
 return 0;
 }
