
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void dot_product(int *d_a, int *d_b,int *d_c){
	int i = threadIdx.x;
	__shared__ int temp[4];
	temp[i] = d_a[i] * d_b[i];
 	__syncthreads();
	if(i==1){
	int sum = 0;
	sum = temp[0]+temp[1]+temp[2]+temp[3];	
	*d_c = sum;	
}
}

int main(void){
 printf("Hello, World - from CPU!\n");
 int a[4] = {22,13,16,5};
 int b[4] = {5,22,17,37};
 int c[0];
 int *d_a; 
 int *d_b;
 int *d_c;
 hipMalloc((void**)&d_a,sizeof(int)*4);
 hipMalloc((void**)&d_b,sizeof(int)*4);
 hipMalloc((void**)&d_c,sizeof(int));
 hipMemcpy(d_a,a,sizeof(int)*4,hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,sizeof(int)*4,hipMemcpyHostToDevice);

 dot_product<<<2,2>>>(d_a,d_b,d_c);
 hipMemcpy(c,d_c,sizeof(int)*1,hipMemcpyDeviceToHost);
 printf("%d",c[0]);
 
 //printf("%d,%d,%d,%d\n",c[0],c[1],c[2],c[3]);
 return 0;
 }
