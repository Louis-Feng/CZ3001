
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_GPU(void){
	int i = threadIdx.x;
	printf("hello from GPU[%d]!\n",i);

}
int main(void){
 printf("Hello, World - from CPU!\n");
 hello_GPU<<<2,3>>>();

 hipDeviceSynchronize();
 return 0;
 }

